/*
 * Perform memory verification and transfer tests on graphics cards supporting 
 * the nVidia CUDA API.
 *
 * Written by Jonathan Coles
 *
 * Build with: make clean && make
 *
 * Last Modified On: 21-DEC-2007
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <cutil.h>
#include "timing.h"

CPUDEFS

#define CHECK(err, msg) if (ret != hipSuccess) { printf msg; goto error; }

hipError_t MALLOC_HOST(void **ptr, size_t size)
{
    *ptr = malloc(size);
    return *ptr != NULL ? hipSuccess : hipErrorOutOfMemory;
}

hipError_t MEMSET_HOST(void *ptr, int val, size_t size) 
{ 
    memset(ptr, val, size); 
    return hipSuccess;
}
hipError_t FREE_HOST(void *ptr)
{ 
    free(ptr);
    return hipSuccess;
}

int verify_copy_tests(struct hipDeviceProp_t *prop)
{
    long i, j, nbytes;
    unsigned char *host = NULL, *dev = NULL;
    hipError_t ret = hipSuccess;
    
    const long ntransfers = 3;
    const long max_nbytes = 2*prop->totalGlobalMem - 1;

    for (nbytes=1; nbytes <= max_nbytes; nbytes *= 2)
    {
        if (nbytes > prop->totalGlobalMem) nbytes = prop->totalGlobalMem;

        printf("Verifying %i byte transfers...\n", nbytes);
        ret = MALLOC_HOST((void **)&host, nbytes); 
        CHECK(ret, ("Can't allocate %i bytes on host\n", nbytes));
        ret = MEMSET_HOST(host, (nbytes & 0xFF), nbytes);
        CHECK(ret, ("Can't set %i bytes on host\n", nbytes));
        ret = hipMalloc((void **)&dev, nbytes);
        CHECK(ret, ("Can't allocate %i bytes on device.\n", nbytes));
        for (j=0; j < ntransfers; j++)
        {
            ret = hipMemcpy(dev, host, nbytes, hipMemcpyHostToDevice);
            CHECK(ret, ("Can't copy %i bytes to device\n", nbytes));
            MEMSET_HOST(host, 0, nbytes); /* In case the memcpy is just failing */
            ret = hipMemcpy(host, dev, nbytes, hipMemcpyDeviceToHost);
            CHECK(ret, ("Can't copy %i bytes to host\n", nbytes));
            for (i=0; i < nbytes; i++) 
            {
                if (host[i] != (nbytes & 0xFF))
                {
                    printf("Mismatched bytes after transfer (%i: %i, %i)\n", 
                           i, host[i], (nbytes & 0xFF));
                    goto error;
                }
            }
        }
        ret = hipFree(dev); dev = NULL; CHECK(ret, (""));
        ret = FREE_HOST(host); dev = NULL; CHECK(ret, (""));
    }

error:

    if (host != NULL) FREE_HOST(host);
    if (dev  != NULL) hipFree(dev);

    if (ret != hipSuccess) printf("CUDA Error: %s\n", hipGetErrorString(ret));

    return ret != hipSuccess;
}

int transfer_rate_tests(struct hipDeviceProp_t *prop)
{
    long j, nbytes;
    unsigned char *host = NULL, *dev = NULL;
    double start, end;
    hipError_t ret;
    
    const long ntransfers = 10000;
    const long max_nbytes = 2*prop->totalGlobalMem - 1;

    for (nbytes=1; nbytes <= max_nbytes; nbytes *= 2)
    {
        if (nbytes > prop->totalGlobalMem) nbytes = prop->totalGlobalMem;

        printf("Timing %i byte transfers... ", nbytes);
        ret = MALLOC_HOST((void **)&host, nbytes); 
        CHECK(ret, ("Can't allocate %i bytes on host\n", nbytes));
        ret = hipMalloc((void **)&dev, nbytes);
        CHECK(ret, ("Can't allocate %i bytes on device.\n", nbytes));
        
        start = CPUTIME;
        for (j=0; j < ntransfers; j++)
            ret = hipMemcpy(dev, host, nbytes, hipMemcpyHostToDevice);
        end = CPUTIME;
        printf("%e ", (end-start) / ntransfers);
        printf("%f ", (double)nbytes / ((end-start) / (double)ntransfers) / (1024*1024*1024.0));

        start = CPUTIME;
        for (j=0; j < ntransfers; j++)
            ret = hipMemcpy(host, dev, nbytes, hipMemcpyDeviceToHost);
        end = CPUTIME;
        printf("%e ", (end-start) / ntransfers);
        printf("%f ", (double)nbytes / ((end-start) / (double)ntransfers) / (1024*1024*1024.0));

        printf("\n");

        hipFree(dev); dev = NULL;
        FREE_HOST(host); host = NULL;
    }

error:

    if (host != NULL) FREE_HOST(host);
    if (dev  != NULL) hipFree(dev);

    return 0;
}

int main(int argc, char **argv)
{
    int i;
    int deviceCount;
    struct hipDeviceProp_t prop;
    hipError_t err;

    hipGetDeviceCount(&deviceCount);

    printf("Found %i device(s).\n", deviceCount);

    for (i=0; i < deviceCount; i++)
    {
        printf("--------------------------------------------\n"
               "Device %i\n", i);

        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess)
        {
            printf("Error %i\n", err);
            continue;
        }

        printf("Name: '%s'\n"
               "Total Global Memory: %u\n"
               "Shared Memory Per Block: %u\n"
               "Registers Per Block: %i\n"
               "Warp Size: %i\n"
               "Memory Pitch: %u\n"
               "Maximum Threads Per Block: %i\n"
               "Maximum Size of Each Block Dimension: %i %i %i\n"
               "Maximum Size of Each Grid Dimension: %i %i %i\n"
               "Total Constant Memory: %i\n"
               "Revision: %i.%i\n"
               "Clockrate: %iHz\n",
               prop.name,
               prop.totalGlobalMem,
               prop.sharedMemPerBlock,
               prop.regsPerBlock,
               prop.warpSize,
               prop.memPitch,
               prop.maxThreadsPerBlock,
               prop.maxThreadsDim[0],
               prop.maxThreadsDim[1],
               prop.maxThreadsDim[2],
               prop.maxGridSize[0],
               prop.maxGridSize[1],
               prop.maxGridSize[2],
               prop.totalConstMem,
               prop.major,
               prop.minor,
               prop.clockRate);
        printf("Size of Property Structure: %u\n", sizeof(prop));

        hipSetDevice(i);
        verify_copy_tests(&prop);
        transfer_rate_tests(&prop);
    }

    return 0;
}

